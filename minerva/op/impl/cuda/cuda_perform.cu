#include "op/impl/cuda/cuda_kernel.h"
#include "op/impl/cuda/cuda_perform.h"
#include "./cuda_perform_system_include.h"
#include <hipblas.h>
#include <hipDNN.h>
#include <hiprand.h>
#include <limits>

namespace minerva {
namespace cuda {

static void FindConfiguration(size_t size, int& num_blocks, int& num_threads) {
  num_threads = 0;
  if(size <= 32)
    num_threads = 32;
  else if(size <= 64)
    num_threads = 64;
  else if(size <= 128)
    num_threads = 128;
  else if(size <= 256)
    num_threads = 256;
  else if(size <= 512)
    num_threads = 512;
  else
    num_threads = 1024;
  num_blocks = static_cast<int>((size + num_threads - 1) / num_threads);
  if (num_blocks < 0 || 128 < num_blocks) {
    num_blocks = 128;
  }
}

void CudaPerformDotMult(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, MultOp());
  CheckCudaError("CudaPerformDotMult");
}

void CudaPerformDotDiv(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, DivOp());
  CheckCudaError("CudaPerformDotDiv");
}

void CudaPerformAdd(float* a, float* b, float* c, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(a, b, c, size, SumOp());
  CheckCudaError("CudaPerformAdd");
  //float one = 1.0;
  //CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  //CUBLAS_CALL(hipblasSaxpy(handle, size, &one, b, 1, c, 1));
}

void CudaPerformCopy(float* a, float* b, size_t size, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, b, 1));
}

void CudaPerformSub(float* a, float* b, float* c, size_t size, hipblasHandle_t handle) {
  float minus_one = -1.0;
  CUBLAS_CALL(hipblasScopy(handle, size, a, 1, c, 1));
  CUBLAS_CALL(hipblasSaxpy(handle, size, &minus_one, b, 1, c, 1));
}

void CudaPerformMatMult(float* a, float* b, float* c, int m, int n, int k, hipblasHandle_t handle) {
  float one = 1.0;
  float zero = 0.0;
  CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &one, a, m, b, k, &zero, c, m));
}

void CudaPerformScale(float* in_data, float* res_data, size_t size, float val, hipblasHandle_t handle) {
  CUBLAS_CALL(hipblasScopy(handle, size, in_data, 1, res_data, 1));
  CUBLAS_CALL(hipblasSscal(handle, size, &val, res_data, 1));
}

void CudaPerformTranspose(float* a, float* c, int m, int n, hipblasHandle_t handle) {
  float zero = 0.0;
  float one = 1.0;
  CUBLAS_CALL(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, &one, a, m, &zero, c, n, c, n));
}

void CudaPerformConstAdd(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, SumOp());
  CheckCudaError("CudaPerformConstAdd");
}

void CudaPerformLeftConstSub(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseSubOp());
  CheckCudaError("CudaPerformLeftConstSub");
}

void CudaPerformLeftConstDiv(float* in, float* out, float val, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, val, size, ReverseDivOp());
  CheckCudaError("CudaPerformLeftConstDiv");
}

void CudaPerformNormAddOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnCol");
}

void CudaPerformNormSubOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnCol");
}

void CudaPerformNormMultOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnCol");
}

void CudaPerformNormDivOnCol(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnColKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnCol");
}

void CudaPerformNormAddOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SumOp());
  CheckCudaError("CudaPerformNormAddOnRow");
}

void CudaPerformNormSubOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, SubOp());
  CheckCudaError("CudaPerformNormSubOnRow");
}

void CudaPerformNormMultOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, MultOp());
  CheckCudaError("CudaPerformNormMultOnRow");
}

void CudaPerformNormDivOnRow(float* matrix, float* row, float* res, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformNormOnRowKernel<<<block, thread, 0, stream>>>(matrix, row, res, m, n, DivOp());
  CheckCudaError("CudaPerformNormDivOnRow");
}

void CudaPerformReductionSumOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnCol");
}

void CudaPerformReductionMaxOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnCol");
}

void CudaPerformReductionSumOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, SumOp());
  CheckCudaError("CudaPerformReductionSumOnRow");
}

void CudaPerformReductionMaxOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformReductionOnRowKernel<<<block, thread, 0, stream>>>(in, out, m, n, MaxOp());
  CheckCudaError("CudaPerformReductionMaxOnRow");
}

void CudaPerformMaxIndexOnCol(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(n, block, thread);
  CudaPerformMaxIndexOnColKernel<<<block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnCol");
}

void CudaPerformMaxIndexOnRow(float* in, float* out, int m, int n, hipStream_t stream) {
  int block, thread;
  FindConfiguration(m, block, thread);
  CudaPerformMaxIndexOnRowKernel << <block, thread, 0, stream>>>(in, out, m, n);
  CheckCudaError("CudaPerformMaxIndexOnRow");
}

void CudaPerformReshape(float* in, float* out, size_t size, hipStream_t stream) {
  CUDA_CALL(hipMemcpyAsync(out, in, size, hipMemcpyDefault, stream));
}

void CudaPerformElewiseExp(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, ExpOp());
  CheckCudaError("CudaPerformEleWiseExp");
}

void CudaPerformElewiseLn(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, LnOp());
  CheckCudaError("CudaPerformEleWiseLn");
}

void CudaPerformElewiseNegative(float* in, float* out, size_t size, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformDotKernel<<<block, thread, 0, stream>>>(in, out, size, NegativeOp());
  CheckCudaError("CudaPerformEleWiseNegative");
}

void CudaPerformConvForward(float* bottom, float* filter, float* bias, float* top, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnTensorDescriptor_t bias_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  float one = 1;
  float zero = 0;
  hipdnnConvolutionFwdAlgo_t algorithm;
  size_t workspace_size;
  void* workspace;
  CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(handle, bottom_desc, filter_desc, conv_desc, top_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algorithm));
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(handle, bottom_desc, filter_desc, conv_desc, top_desc, algorithm, &workspace_size));
  CUDA_CALL(hipMalloc(&workspace, workspace_size));
  CUDNN_CALL(hipdnnConvolutionForward(handle, &one, bottom_desc, bottom, filter_desc, filter, conv_desc, algorithm, workspace, workspace_size, &zero, top_desc, top));
  CUDNN_CALL(hipdnnAddTensor(handle, CUDNN_ADD_SAME_C, &one, bias_desc, bias, &one, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDA_CALL(hipFree(workspace));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bias_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformConvBackwardData(float* top_diff, float* filter, float* bottom_diff, int num_images, int bottom_num_channels, int top_num_channels, int top_height, int top_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t bottom_diff_desc;
  hipdnnFilterDescriptor_t filter_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnTensorDescriptor_t top_diff_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_diff_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, (top_height - 1) * stride_vertical + filter_height - 2 * pad_height, (top_width - 1) * stride_horizontal + filter_width - 2 * pad_width));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnConvolutionBackwardData(handle, &one, filter_desc, filter, top_diff_desc, top_diff, conv_desc, &zero, bottom_diff_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_diff_desc));
}

void CudaPerformConvBackwardFilter(float* bottom, float* top_diff, float* filter_diff, int num_images, int bottom_num_channels, int top_num_channels, int bottom_height, int bottom_width, int pad_height, int pad_width, int stride_vertical, int stride_horizontal, int filter_height, int filter_width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnFilterDescriptor_t filter_diff_desc;
  hipdnnConvolutionDescriptor_t conv_desc;
  hipdnnTensorDescriptor_t top_diff_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filter_diff_desc));
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, bottom_num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filter_diff_desc, HIPDNN_DATA_FLOAT, top_num_channels, bottom_num_channels, filter_height, filter_width));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(conv_desc, pad_height, pad_width, stride_vertical, stride_horizontal, 1, 1, HIPDNN_CONVOLUTION));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, (bottom_height + 2 * pad_height - filter_height) / stride_vertical + 1, (bottom_width + 2 * pad_width - filter_width) / stride_horizontal + 1));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnConvolutionBackwardFilter(handle, &one, bottom_desc, bottom, top_diff_desc, top_diff, conv_desc, &zero, filter_diff_desc, filter_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filter_diff_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformConvBackwardBias(float* top_diff, float* bias_diff, int num_images, int top_num_channels, int top_height, int top_width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t bias_diff_desc;
  hipdnnTensorDescriptor_t top_diff_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bias_diff_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_diff_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bias_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, top_num_channels, 1, 1));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_diff_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, top_num_channels, top_height, top_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnConvolutionBackwardBias(handle, &one, top_diff_desc, top_diff, &zero, bias_diff_desc, bias_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_diff_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bias_diff_desc));
}

void CudaPerformInstanceSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformChannelSoftmaxForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxForward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformInstanceSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, &one, desc, top, desc, diff, &zero, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformChannelSoftmaxBackward(float* diff, float* top, float* bottom, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnSoftmaxBackward(handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &one, desc, top, desc, diff, &zero, desc, bottom));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformSigmoidForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_SIGMOID, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformReluForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_RELU, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformTanhForward(float* bottom, float* top, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationForward(handle, HIPDNN_ACTIVATION_TANH, &one, desc, bottom, &zero, desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformSigmoidBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_SIGMOID, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformReluBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_RELU, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformTanhBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int height, int width, hipStream_t stream, hipdnnHandle_t handle) {
  hipdnnTensorDescriptor_t desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, height, width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnActivationBackward(handle, HIPDNN_ACTIVATION_TANH, &one, desc, top, desc, top_diff, desc, bottom, &zero, desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(desc));
}

void CudaPerformMaxPoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, &one, bottom_desc, bottom, &zero, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingForward(float* bottom, float* top, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingForward(handle, pool_desc, &one, bottom_desc, bottom, &zero, top_desc, top));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformMaxPoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_MAX, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, &one, top_desc, top, top_desc, top_diff, bottom_desc, bottom, &zero, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformAveragePoolingBackward(float* bottom, float* top, float* top_diff, float* bottom_diff, int num_images, int num_channels, int bottom_height, int bottom_width, int stride_vertical, int stride_horizontal, int window_height, int window_width, int pad_height, int pad_width, hipStream_t stream, hipdnnHandle_t handle) {
  // Calculate the dimension after pooling
  int pooled_height = (bottom_height + 2 * pad_height - window_height + stride_vertical - 1) / stride_vertical + 1;
  if (0 <= (pooled_height - 1) * stride_vertical - bottom_height - pad_height) {
    --pooled_height;
  }
  int pooled_width = (bottom_width + 2 * pad_width - window_width + stride_horizontal - 1) / stride_horizontal + 1;
  if (0 <= (pooled_width - 1) * stride_horizontal - bottom_width - pad_width) {
    --pooled_width;
  }
  hipdnnTensorDescriptor_t bottom_desc;
  hipdnnPoolingDescriptor_t pool_desc;
  hipdnnTensorDescriptor_t top_desc;

  CUDNN_CALL(hipdnnCreateTensorDescriptor(&bottom_desc));
  CUDNN_CALL(hipdnnCreatePoolingDescriptor(&pool_desc));
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&top_desc));

  CUDNN_CALL(hipdnnSetTensor4dDescriptor(bottom_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, bottom_height, bottom_width));
  CUDNN_CALL(hipdnnSetPooling2dDescriptor(pool_desc, HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, window_height, window_width, pad_height, pad_width, stride_vertical, stride_horizontal));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(top_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, num_images, num_channels, pooled_height, pooled_width));

  float one = 1;
  float zero = 0;
  CUDNN_CALL(hipdnnPoolingBackward(handle, pool_desc, &one, top_desc, top, top_desc, top_diff, bottom_desc, bottom, &zero, bottom_desc, bottom_diff));
  CUDA_CALL(hipStreamSynchronize(stream));  // Synchronize before destruction

  CUDNN_CALL(hipdnnDestroyTensorDescriptor(top_desc));
  CUDNN_CALL(hipdnnDestroyPoolingDescriptor(pool_desc));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(bottom_desc));
}

void CudaPerformRandn(float* dst, size_t size, unsigned int seed, float mean, float var) {
  hiprandGenerator_t gen;
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
  CURAND_CALL(hiprandGenerateNormal(gen, dst, size, mean, var));
  CURAND_CALL(hiprandDestroyGenerator(gen));
}

void CudaPerformRandBernoulli(float* dst, size_t size, unsigned int seed, float p, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformRandBernoulliKernel<<<block, thread, 0, stream>>>(dst, size, seed, p);
# if defined(_MSC_VER)
  CheckCudaError(__FUNCTION__);
# else
  CheckCudaError(__func__);
# endif
}

void CudaPerformFill(float* dst, size_t size, float val, hipStream_t stream) {
  int block, thread;
  FindConfiguration(size, block, thread);
  CudaPerformFillKernel<<<block, thread, 0, stream>>>(dst, size, val);
  CheckCudaError("CudaPerformFill");
}

void CudaPerformLRNForward(float* bottom, float* scale, float* res, int local_size, float alpha, float beta, int num_img, int channel, int width, int height, hipStream_t stream) {
  int block, thread, size;
  size = num_img * height * width;
  FindConfiguration(size, block, thread);
  LRNFillScale<<<block, thread, 0, stream>>>(
      size, bottom, num_img, channel, height, width, local_size,
      alpha / local_size, scale);
  CheckCudaError("LRNFillScale");

  size = num_img * channel * width * height;
  FindConfiguration(size, block, thread);
  // NOLINT_NEXT_LINE(whitespace/operators)
  LRNComputeOutput<<<block, thread, 0, stream>>>(size, bottom, scale, -beta, res);
  CheckCudaError("LRNComputeOutput");
}

void CudaPerformLRNBackward(float* bottom_data, float* top_data, float* scale, float* top_diff, float* bottom_diff, int local_size, float alpha, float beta, int num_img, int channel, int width, int height, hipStream_t stream) {
  int block, thread;
  int size = num_img * width * height;
  FindConfiguration(size, block, thread);
  LRNComputeDiff<<<block, thread, 0, stream>>>(
      size, bottom_data, top_data, scale, top_diff,  num_img, channel, height, width, local_size,
      -beta, float(2. * alpha * beta / local_size), bottom_diff);
  CheckCudaError("LRNBackward");
}

void CudaPerformSelect(float* dst, float* src, std::vector<int> indices, size_t cols, size_t rows, hipStream_t stream) {
  int block, thread;
  int size = cols * rows;
  int* indices_ptr;
  FindConfiguration(size, block, thread);
  CUDA_CALL(hipMalloc(&indices_ptr, indices.size() * sizeof(int)));
  CUDA_CALL(hipMemcpyAsync(indices_ptr, &indices[0], indices.size() * sizeof(int), hipMemcpyDefault, stream));
  SelectKernel<<<block, thread, 0, stream>>>(dst, src, &indices[0], cols, rows, indices.size());
  CheckCudaError("Select");
}

}  // namespace cuda
}  // namespace minerva

